
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include <stdlib.h>
#include <unistd.h>

static int SLEEP_TIME = 50000;
static int GENERATION_STEP = 1;

__global__ void singleBlockLifeKernel(uint32_t *cols, int numGenerations) {
  __shared__ uint8_t grid[1024]; // TODO Should this be uint32_t?

  int colIdx = threadIdx.x;

  // Copy data from global memory to shared memory
  uint32_t colData = cols[colIdx];

  // Split the data out into an easy to handle array
  for (int i = 0; i < 32; ++i) {
    grid[i * 32 + colIdx] = ((colData & 1 << i)) >> i;
  }

  // The bit mask is a quick and dirty way of computing the positive bounded %32
  uint8_t leftIdx = ((colIdx - 1) & 0x1f);
  uint8_t rightIdx = ((colIdx + 1) & 0x1f);

  for (int g = 0; g < numGenerations; ++g) {
    uint8_t lastSides = 0, lastMiddle = 0, thisSides = 0, thisMiddle = 0,
            nextSides = 0, nextMiddle = 0;

    // Get the nieghbors from the row below
    lastSides = grid[31 * 32 + leftIdx] & 1;
    lastSides += grid[31 * 32 + rightIdx] & 1;
    lastMiddle = grid[31 * 32 + colIdx];

    // Get the neighbors in this row and the cell itself
    thisSides = grid[leftIdx] & 1;
    thisSides += grid[rightIdx] & 1;
    thisMiddle = grid[colIdx];

    // Perform cellular automata
    for (int i = 0; i < 31; ++i) {

      // Get the neighbors in the next row
      nextSides = grid[(i + 1) * 32 + leftIdx] & 1;
      nextSides += grid[(i + 1) * 32 + rightIdx] & 1;
      nextMiddle = grid[(i + 1) * 32 + colIdx];

      // Calculate the numbers of neighbors still alive
      uint8_t neighbors =
          lastSides + lastMiddle + thisSides + nextSides + nextMiddle;

      // Write the next state directly to the memory location already allocated
      // for this square, just in a differnt bit
      // TODO Maybe just make this a macro?
      grid[i * 32 + colIdx] |=
          (~neighbors >> 1 & neighbors & (thisMiddle | neighbors) << 1) & 2;

      // The current row becomes the last row, mutatis mutandis for the next row
      lastSides = thisSides;
      lastMiddle = thisMiddle;
      thisSides = nextSides;
      thisMiddle = nextMiddle;
    }

    // The next row for the last row in the cell will be the dame as the first
    // row
    nextSides = grid[leftIdx] & 1;
    nextSides += grid[rightIdx] & 1;
    nextMiddle = grid[colIdx] & 1;

    // Compute the number of neighbors for this row
    uint8_t neighbors =
        lastSides + lastMiddle + thisSides + nextSides + nextMiddle;

    // Write the next state directly to the memory location already allocated
    // for this square, just in a differnt bit
    grid[31 * 32 + colIdx] |=
        (~neighbors >> 1 & neighbors & (thisMiddle | neighbors) << 1) & 2;

    // Make sure all threads have finished the current generation before starting the next generation
    __syncthreads();

    // Shift the next state of the cell into the current state of the cell
    for (int i = 0; i < 32; ++i) {
      grid[i * 32 + colIdx] >>= 1;
    }
  }

  // Clear the register to store compacted data
  colData = 0;

  // Cram the data back into a single value
  for (int i = 0; i < 32; ++i) {
    colData |= ((grid[i * 32 + colIdx]) & 1) << i;
  }

  // Copy the data back into global memory
  cols[colIdx] = colData;
}

void generateGrid(uint32_t *&cols) {

  uint32_t seed = std::chrono::duration_cast<std::chrono::milliseconds>(
                      std::chrono::system_clock::now().time_since_epoch())
                      .count();
  srand(seed);

  for (int i = 0; i < 32; ++i) {
    cols[i] = rand() & rand() & 0xFFFFFFFF;
  }
}

void drawGrid(uint32_t *col, int generation) {
  printf("\033[H");
  for (int y = 0; y < 32; ++y) {
    // printf("\n\033[1;%dH", y+1);
    for (int x = 0; x < 32; ++x)
      printf((col[x] & (1l << y)) ? "██" : "  ");
    printf("\n");
  }
  printf("%d    ", generation);
  usleep(SLEEP_TIME);
}

int main(int argc, char **argv) {

  if (argc > 1)
    GENERATION_STEP = std::stoi(argv[1]);

  if (argc > 2)
    SLEEP_TIME = std::stoi(argv[2]);

  uint32_t *cols;
  uint32_t generation = 0;

  hipMallocManaged(&cols, sizeof(uint32_t) * 32);

  generateGrid(cols);
  drawGrid(cols, generation);

  while (true) {
    singleBlockLifeKernel<<<1, 32>>>(cols, GENERATION_STEP);
    generation += GENERATION_STEP;
    hipDeviceSynchronize();
    drawGrid(cols, generation);
  }
  hipFree(cols);
  return 0;
}