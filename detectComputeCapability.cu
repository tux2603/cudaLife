
#include <hip/hip_runtime.h>
#include <iostream>

int main() {
  hipDevice_t dev;
  hipDeviceGet(&dev, 0);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  printf("%d%d\n", deviceProp.major, deviceProp.minor);
}